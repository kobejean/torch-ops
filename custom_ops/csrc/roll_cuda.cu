#include "hip/hip_runtime.h"
#include "registration.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

namespace custom_ops {
namespace tensor {

// Constant memory for dimension info
// Max 8 dimensions should cover most use cases
constexpr int MAX_ROLL_DIMS = 8;
__constant__ int64_t c_sizes[MAX_ROLL_DIMS];
__constant__ int64_t c_strides[MAX_ROLL_DIMS];
__constant__ int64_t c_shifts[MAX_ROLL_DIMS];

template <typename scalar_t>
__global__ void roll_kernel(
    const scalar_t* __restrict__ input,
    scalar_t* __restrict__ output,
    const int64_t n_elem,
    const int n_dims
) {
    const int64_t flat_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (flat_idx >= n_elem) return;
    
    int64_t offset = 0;
    
    #pragma unroll
    for (int d = 0; d < n_dims; ++d) {
        const int64_t old_idx = (flat_idx / c_strides[d]) % c_sizes[d];
        const int64_t new_idx = (old_idx + c_shifts[d]) % c_sizes[d];
        offset += (new_idx - old_idx) * c_strides[d];
    }
    
    output[flat_idx + offset] = input[flat_idx];
}

at::Tensor roll_cuda(
    const at::Tensor& input,
    at::IntArrayRef shifts,
    at::IntArrayRef dims
) {
    TORCH_CHECK(input.device().is_cuda(), "Input tensor must be on CUDA");
    
    // Handle empty tensor
    if (input.numel() == 0) {
        return input.clone();
    }
    
    // Validate dimensions
    const int64_t ndim = input.ndimension();
    TORCH_CHECK(shifts.size() == dims.size(),
                "shifts and dims must have the same length");
    
    // Create a map of dimension to shift amount
    std::vector<int64_t> shift_amounts(ndim, 0);
    for (size_t i = 0; i < dims.size(); ++i) {
        int64_t dim = dims[i];
        // Handle negative dims
        if (dim < 0) {
            dim += ndim;
        }
        TORCH_CHECK(dim >= 0 && dim < ndim,
                    "Dimension out of range (expected to be in range of [",
                    -ndim, ", ", ndim - 1, "], but got ", dims[i], ")");
        
        // Accumulate shifts for the same dimension
        shift_amounts[dim] += shifts[i];
    }
    
    // Convert shifts to positive values and filter out zero shifts
    auto sizes = input.sizes();
    auto strides = input.strides();
    
    std::vector<int64_t> eff_sizes;
    std::vector<int64_t> eff_strides;
    std::vector<int64_t> eff_shifts;
    
    for (int64_t d = 0; d < ndim; ++d) {
        if (sizes[d] > 0) {
            // Convert negative shifts to positive equivalents
            int64_t normalized_shift = ((shift_amounts[d] % sizes[d]) + sizes[d]) % sizes[d];
            
            // Only include dimensions with non-zero shifts
            if (normalized_shift != 0) {
                eff_sizes.push_back(sizes[d]);
                eff_strides.push_back(strides[d]);
                eff_shifts.push_back(normalized_shift);
            }
        }
    }
    
    const int eff_dims = eff_sizes.size();
    
    // Early exit if no dimensions need shifting
    if (eff_dims == 0) {
        return input.clone();
    }
    
    // Check dimension limit for constant memory
    TORCH_CHECK(eff_dims <= MAX_ROLL_DIMS, 
                "Too many effective dimensions (", eff_dims, ") for roll operation. Max supported: ", MAX_ROLL_DIMS);
    
    // Copy dimension data to constant memory (much faster than global memory)
    hipMemcpyToSymbol(HIP_SYMBOL(c_sizes), eff_sizes.data(), eff_dims * sizeof(int64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(c_strides), eff_strides.data(), eff_dims * sizeof(int64_t));
    hipMemcpyToSymbol(HIP_SYMBOL(c_shifts), eff_shifts.data(), eff_dims * sizeof(int64_t));
    
    // Create output tensor
    auto output = torch::empty_like(input);
    const int64_t num_elements = input.numel();
    
    // Launch kernel
    const int threads = 256;
    const int blocks = (num_elements + threads - 1) / threads;
    
    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "roll_cuda", [&] {
        roll_kernel<scalar_t><<<blocks, threads>>>(
            input.data_ptr<scalar_t>(),
            output.data_ptr<scalar_t>(),
            num_elements,
            eff_dims
        );
    });
    
    // Check for CUDA errors
    hipError_t error = hipGetLastError();
    TORCH_CHECK(error == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(error));
    
    return output;
}

}} // namespace custom_ops::tensor