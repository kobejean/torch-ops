#include "hip/hip_runtime.h"
#include "registration.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace custom_ops {
namespace fused {

__global__ void mul_add_relu_kernel(
    const float* x,
    const float* weight,
    const float* bias,
    float* output,
    int64_t size
) {
    int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float result = x[idx] * weight[idx] + bias[idx];
        output[idx] = fmaxf(0.0f, result);  // ReLU
    }
}

at::Tensor mul_add_relu_cuda(const at::Tensor& x, const at::Tensor& weight, const at::Tensor& bias) {
    utils::check_cuda_tensors({x, weight, bias});
    utils::check_same_size(x, weight);
    utils::check_same_size(x, bias);
    TORCH_CHECK(x.dtype() == at::kFloat, "All tensors must be float32");
    TORCH_CHECK(weight.dtype() == at::kFloat, "All tensors must be float32");
    TORCH_CHECK(bias.dtype() == at::kFloat, "All tensors must be float32");
    
    auto output = torch::empty_like(x);
    const int64_t size = x.numel();
    
    const int threads = 256;
    const int blocks = (size + threads - 1) / threads;
    
    mul_add_relu_kernel<<<blocks, threads>>>(
        x.data_ptr<float>(),
        weight.data_ptr<float>(),
        bias.data_ptr<float>(),
        output.data_ptr<float>(),
        size
    );
    
    hipError_t error = hipGetLastError();
    TORCH_CHECK(error == hipSuccess, "CUDA kernel failed: ", hipGetErrorString(error));
    
    return output;
}

}} // namespace custom_ops::fused